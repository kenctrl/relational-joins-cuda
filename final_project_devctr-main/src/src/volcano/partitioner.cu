#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "cassert"

template <int radix_size, int block>
struct cache_t {
    int shuffle[radix_size * block];
    int value[radix_size * block];
    int end[radix_size];
    int start[radix_size];
    int head[radix_size];
};

// assumes mask == threadIdx.x
// first 32 bits of head is the size of the bucket
// next 32 bits of head in the position of the head
template<int radix_size, int shift, int block, int32_t bucket_size>
__global__ void first_pass(int* __restrict__ num_buckets, 
                           unsigned long long* __restrict__ head,
                           int * __restrict__ part,
                           int * __restrict__ idx,
                           int * __restrict__ value,
                           int * __restrict__ cnt,
                           const int* __restrict__ keys, 
                           int num_rows) {
    __shared__ cache_t<radix_size, block> cache;
    const int start = blockIdx.x * blockDim.x * block;
    const int tid = threadIdx.x;
    int regs[block];
    cache.end[tid] = 0;
    __syncthreads();

    #pragma unroll
    for(int k = 0; k < block; k++) {
        const int i = start + k * blockDim.x + tid;
        if(i < num_rows) {
            regs[k] = keys[i];
            int idx = (regs[k] >> shift) & (radix_size - 1);
            atomicAdd(&cache.end[idx], 1);
        }
    }
    __syncthreads();
    if(tid == 0) {
        int sum = 0;
        for(int i = 0; i < radix_size; i++) {
            int cnt = cache.end[i];
            cache.start[i] = sum;
            cache.head[i] = sum;
            sum += cnt;
            cache.end[i] = sum;
        }
    }
    __syncthreads();

    #pragma unroll
    for(int k = 0; k < block; k++) {
        const int i = start + k * blockDim.x + tid;
        if(i < num_rows) {
            const int idx = (regs[k] >> shift) & (radix_size - 1);
            int cur = atomicAdd(&cache.head[idx], 1);
            cache.shuffle[cur] = i;
            cache.value[cur] = regs[k];
        }
    }

    __syncthreads();

    int left = cache.start[tid];
    int right = cache.end[tid];
    unsigned long long length = right - left;
    while(length > 0) {
        atomicMin(&head[tid], ((unsigned long long) bucket_size) << 32);
        unsigned long long prev = atomicAdd(&head[tid], length << 32);

        const uint32_t len = prev >> 32;
        const uint32_t start = prev & 0xFFFFFFFF;
        const uint32_t cur_bucket = start / bucket_size;

        uint32_t run_length = 0;
        if(len < bucket_size) {
            if(len + length < bucket_size) {
                run_length = length;
            } else {
                run_length = bucket_size - len;

                int next_bucket = atomicAdd(num_buckets, 1);

                unsigned long long new_start = next_bucket * bucket_size;
                atomicExch(&head[tid], new_start);

                part[next_bucket] = tid;
            }
        }
        atomicAdd(&cnt[cur_bucket], run_length);

        for(int x = 0; x < run_length; x++) {
            idx[start + len + x] = cache.shuffle[left + x];
            value[start + len + x] = cache.value[left + x];
        }
        left += run_length;
        length -= run_length;
    }
}

template <int radix_size, int bucket_size>
__global__ void init_buckets(int *num_buckets,
                             unsigned long long *head,
                             int* part,
                             int* cnt,
                             int max_buckets) {
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i == 0) {
        *num_buckets = radix_size;
    }
    if(i < max_buckets) {
        part[i] = (i < radix_size) ? i : -1;
        cnt[i] = 0;
    }
    if(i < radix_size) {
        head[i] = i * bucket_size;
    }
}


// returns the cpu pointer
int* launch_first_pass(const int* keys, int num_rows) {
    constexpr int radix_size = 256;
    constexpr int shift = 1;
    constexpr int bucket_size = 1024;
    const int max_buckets = num_rows / bucket_size + radix_size + 2;

    int *d_keys;
    int *num_buckets;
    unsigned long long *head;
    int *part;
    int *value, *idx;
    int *cnt;

    allocate_mem(&d_keys, false, sizeof(int) * num_rows);
    allocate_mem(&num_buckets, false, sizeof(int));
    allocate_mem(&head, false, sizeof(unsigned long long) * radix_size);
    allocate_mem(&part, false, sizeof(int) * max_buckets);
    allocate_mem(&idx, false, sizeof(int) * max_buckets * bucket_size);
    allocate_mem(&value, false, sizeof(int) * max_buckets * bucket_size);
    allocate_mem(&cnt, false, sizeof(int) * max_buckets);

    CHECK_CUDA_ERROR(
        hipMemcpy(d_keys, keys, sizeof(int) * num_rows, hipMemcpyHostToDevice)
    );

    init_buckets<radix_size, bucket_size><<<max_buckets / 32 + 1, 32>>>(
        num_buckets,
        head,
        part,
        cnt,
        max_buckets
    );
    printf("hello everyone\n");

    auto launch = [&] () {
        constexpr int block = 8;
        const int gridDimX = (num_rows + radix_size * block - 1) / (radix_size * block);
        first_pass<radix_size, shift, block, bucket_size><<<gridDimX, radix_size>>> (
            num_buckets,
            head,
            part,
            idx,
            value,
            cnt,
            d_keys,
            num_rows
        );
        CHECK_LAST_CUDA_ERROR();
    };
    float t = 0;
    SETUP_TIMING()
    TIME_FUNC(launch(), t)
    std::cout << "First pass timing " << t << std::endl;

    int *h_part, *h_idx, *h_cnt, *h_num_buckets;

    CHECK_CUDA_ERROR(
        hipHostMalloc(&h_num_buckets, sizeof(int))
    );

    CHECK_CUDA_ERROR(
        hipHostMalloc(&h_part, sizeof(int) * max_buckets)
    );

    CHECK_CUDA_ERROR(
        hipHostMalloc(&h_cnt, sizeof(int) * max_buckets)
    );

    CHECK_CUDA_ERROR(
        hipHostMalloc(&h_idx, sizeof(int) * max_buckets * bucket_size)
    );

    CHECK_CUDA_ERROR(
        hipMemcpy(h_num_buckets, num_buckets, sizeof(int), hipMemcpyDeviceToHost)
    );

    CHECK_CUDA_ERROR(
        hipMemcpy(h_part, part, sizeof(int) * max_buckets, hipMemcpyDeviceToHost)
    );

    CHECK_CUDA_ERROR(
        hipMemcpy(h_cnt, cnt, sizeof(int) * max_buckets, hipMemcpyDeviceToHost)
    );

    CHECK_CUDA_ERROR(
        hipMemcpy(h_idx, idx, sizeof(int) * max_buckets * bucket_size, hipMemcpyDeviceToHost)
    );

    std::cout << "All memory moved the host" << std::endl;

    std::cout << "max buckets " << max_buckets << std::endl;
    std::cout << "buckets needed " << *h_num_buckets << std::endl;

    int num_bucks = 0;
    for(int i = 0; i < max_buckets; i++) {
        if(h_part[i] != -1) {
            int partition = h_part[i];
            for(int j = 0; j < h_cnt[i]; j++) {
                int index = h_idx[i * bucket_size + j];
                int value = keys[index];
                int recalc = (value >> shift) & (radix_size - 1);
                if(recalc != partition) {
                    std::cout << "bucket no " << i << std::endl;
                    std::cout << "partition " << partition << std::endl;
                    std::cout << "Incorrect at " << index << std::endl;
                    for(int k = 0; k < h_cnt[i]; k++) {
                        std::cout << h_idx[i * bucket_size + k] << " ";
                    }
                    std::cout << std::endl;
                    assert(false);
                }
            }
            ++num_bucks;
        }
    }
    std::cout << num_bucks << std::endl;
    return nullptr;
}

int main() {
    const int num_rows = 1 << 25;
    int *keys;
    CHECK_CUDA_ERROR(
        hipHostMalloc(&keys, sizeof(int) * num_rows)
    );
    for(int i = 0; i < num_rows; i++) {
        keys[i] = rand();
    }
    int *cnt = launch_first_pass(keys, num_rows);
    return 0;
}